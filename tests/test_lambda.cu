#include "hip/hip_runtime.h"
/*
 * test_lambda.cu
 *
 *  Created on: Mar 28, 2012
 *      Author: Filippo Squillace
 */


#include <iostream>
#include <sstream>
#include <string.h>


#include <cusp/csr_matrix.h>
#include <cusp/io/matrix_market.h>
#include <string.h>
#include <cusp/print.h>
#include <cusp/multiply.h>
#include <cusp/transpose.h>
#include <cusp/array1d.h>
#include <cusp/array2d.h>
#include <cusp/krylov/arnoldi.h>
#include <cusp/detail/matrix_base.h>
//#include "../../cusp/krylov/arnoldi.h"


#include <lambda/composite_matrix.h>


#include <cppunit/ui/text/TestRunner.h>
#include <cppunit/TestFixture.h>
#include <cppunit/TestCaller.h>
#include <cppunit/TestSuite.h>
#include <cppunit/extensions/HelperMacros.h>
#include <cppunit/extensions/TestFactoryRegistry.h>


void checkStatus(culaStatus status)
{
	if(!status)
		return;
	if(status == culaArgumentError)
		printf("Invalid value for parameter %d\n", culaGetErrorInfo());
	else if(status == culaDataError)
		printf("Data error (%d)\n", culaGetErrorInfo());
	else if(status == culaBlasError)
		printf("Blas error (%d)\n", culaGetErrorInfo());
	else if(status == culaRuntimeError)
		printf("Runtime error (%d)\n", culaGetErrorInfo());
	else
		printf("%s\n", culaGetStatusString(status));

	culaShutdown();
	exit(EXIT_FAILURE);
}


class LambdaTestCase : public CppUnit::TestFixture {

	CPPUNIT_TEST_SUITE (LambdaTestCase);
	CPPUNIT_TEST (test_host_arnoldi);
	CPPUNIT_TEST_SUITE_END ();

	typedef int    IndexType;
	typedef float ValueType;
	typedef cusp::array2d<float,cusp::device_memory, cusp::column_major> DeviceMatrix_array2d;
	typedef cusp::array2d<float, cusp::host_memory, cusp::column_major>   HostMatrix_array2d;

	typedef cusp::array1d<float,cusp::device_memory> DeviceVector_array1d;
	typedef cusp::array1d<float, cusp::host_memory>   HostVector_array1d;

	typedef cusp::csr_matrix<IndexType, ValueType, cusp::host_memory>   HostMatrix_csr;
	typedef cusp::csr_matrix<IndexType, ValueType, cusp::device_memory>   DeviceMatrix_csr;

	typedef lambda::composite_matrix<IndexType, ValueType, cusp::host_memory, HostMatrix_csr>   HostMatrix_comp;
	typedef lambda::composite_matrix<IndexType, ValueType, cusp::device_memory,  DeviceMatrix_csr> DeviceMatrix_comp;


private:
	DeviceMatrix_comp dev_mat;
	HostMatrix_comp host_mat;


public:

	void setUp()
	{

		culaStatus status;
		status = culaInitialize();
		checkStatus(status);

		std::string path = "data/positive-definite/lehmer20.mtx";


		HostMatrix_csr M11, M12, L11, L21, L22;
		cusp::io::read_matrix_market_file(M11, path);
		cusp::io::read_matrix_market_file(M12, path);
		cusp::io::read_matrix_market_file(L11, path);
		cusp::io::read_matrix_market_file(L21, path);
		cusp::io::read_matrix_market_file(L22, path);

		host_mat = HostMatrix_comp(M11,M12,L11,L21,L22);

		dev_mat = DeviceMatrix_comp(host_mat);

	}

	void tearDown()
	{
		culaShutdown();
	}

	void test_host_multiply(){

	}

	void test_host_arnoldi()
	{




		// TODO A = L11^{-1}*(M_{11}+M_{12}*L_{22}^{-1}*_{21})
		HostMatrix_array2d A;




//		for(size_t i=0; i<path_def_pos.size(); i++){
//
//			size_t m = 10;
//			HostMatrix_array2d H(m, m);
//			HostMatrix_array2d V(host_mat_def_pos[i].num_rows, m);
//			HostVector_array1d f(host_mat_def_pos[i].num_rows, ValueType(0));
//
//			cusp::krylov::arnoldi(host_mat_def_pos[i], H, V, f, 0, 3);
//			cusp::krylov::arnoldi(host_mat_def_pos[i], H, V, f, 2, 5);
//			cusp::krylov::arnoldi(host_mat_def_pos[i], H, V, f, 4, m);
//
//			HostMatrix_array2d A2d;
//			HostMatrix_array2d V2;
//			HostMatrix_array2d H2;
//
//			HostMatrix_array2d C;
//			HostMatrix_array2d C2;
//
//
//			size_t N = host_mat_def_pos[i].num_rows;
//
//			cusp::convert(host_mat_def_pos[i], A2d);
//
//
//			// create submatrix V2
//			cusp::copy(V, V2);
//			V2.resize(N,m);
//
//			// create submatrix H2
//			H2.resize(m,m);
//			size_t l = H.num_rows;
//			for(size_t j=0; j<m; j++)
//				thrust::copy(H.values.begin()+ l*j, H.values.begin()+ l*j +m, H2.values.begin()+ m*j);
//
//			cusp::multiply(A2d, V2, C);
//
//			cusp::multiply(V2, H2, C2);
//
//			cusp::blas::axpy(f.begin() , f.end(), C2.values.begin()+(m-1)*N, ValueType(1));
//
//
//			ValueType errRel = nrmVector("host_arnoldi: "+path_def_pos[i], C.values, C2.values);
//			CPPUNIT_ASSERT( errRel < 1.0e-5 );
//
//		}
	}



	template <typename Array1d>
	ValueType nrmVector(std::string title, Array1d& A, Array1d& A2){
		ValueType nrmA = cusp::blas::nrm2(A);
		ValueType nrmA2 = cusp::blas::nrm2(A2);
		// Calculates the difference and overwrite the matrix C
		cusp::blas::axpy(A, A2, ValueType(-1));
		ValueType nrmDiff = cusp::blas::nrm2(A2);



		ValueType errRel = ValueType(0);
		if(nrmA==ValueType(0))
			errRel = ValueType(1.0e-30);
		else
			errRel = nrmDiff/nrmA;

#ifdef VERBOSE
#ifndef VVERBOSE
		if(errRel != errRel || errRel >= 1.0e-2){ // Checks if error is nan
#endif VVERBOSE

			std::cout << title << ": AbsoluteErr=" << nrmDiff <<\
					" RelativeErr=" << errRel << "\n" << std::endl;
#ifndef VVERBOSE
		}
#endif VVERBOSE
#endif


		return errRel;
	}



};





CPPUNIT_TEST_SUITE_REGISTRATION( LambdaTestCase );

int main(int argc, char** argv)
{

	CppUnit::TextUi::TestRunner runner;
	CppUnit::TestFactoryRegistry &registry = CppUnit::TestFactoryRegistry::getRegistry();
	runner.addTest( registry.makeTest() );
	runner.run();
	return 0;

}



