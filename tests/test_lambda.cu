#include "hip/hip_runtime.h"
/*
 * test_lambda.cu
 *
 *  Created on: Mar 28, 2012
 *      Author: Filippo Squillace
 */


#include <iostream>
#include <sstream>
#include <string.h>


#include <cusp/csr_matrix.h>
#include <cusp/io/matrix_market.h>
#include <cusp/print.h>
#include <cusp/multiply.h>
#include <cusp/transpose.h>
#include <cusp/array1d.h>
#include <cusp/array2d.h>
#include <cusp/krylov/arnoldi.h>
#include <cusp/detail/matrix_base.h>
//#include "../../cusp/krylov/arnoldi.h"


#include <lambda/composite_matrix.h>

#include <cuspla.cu>


#include <cppunit/ui/text/TestRunner.h>
#include <cppunit/TestFixture.h>
#include <cppunit/TestCaller.h>
#include <cppunit/TestSuite.h>
#include <cppunit/extensions/HelperMacros.h>
#include <cppunit/extensions/TestFactoryRegistry.h>


void checkStatus(culaStatus status)
{
	if(!status)
		return;
	if(status == culaArgumentError)
		printf("Invalid value for parameter %d\n", culaGetErrorInfo());
	else if(status == culaDataError)
		printf("Data error (%d)\n", culaGetErrorInfo());
	else if(status == culaBlasError)
		printf("Blas error (%d)\n", culaGetErrorInfo());
	else if(status == culaRuntimeError)
		printf("Runtime error (%d)\n", culaGetErrorInfo());
	else
		printf("%s\n", culaGetStatusString(status));

	culaShutdown();
	exit(EXIT_FAILURE);
}


class LambdaTestCase : public CppUnit::TestFixture {

	CPPUNIT_TEST_SUITE (LambdaTestCase);
	CPPUNIT_TEST (test_host_multiply);
	CPPUNIT_TEST (test_device_multiply);
	CPPUNIT_TEST(test_host_arnoldi);
	CPPUNIT_TEST(test_device_arnoldi);
	CPPUNIT_TEST(test_host_iram);
	CPPUNIT_TEST_SUITE_END ();

	typedef int    IndexType;
	typedef float ValueType;
	typedef cusp::array2d<float,cusp::device_memory, cusp::column_major> DeviceMatrix_array2d;
	typedef cusp::array2d<float, cusp::host_memory, cusp::column_major>   HostMatrix_array2d;

	typedef cusp::array1d<float,cusp::device_memory> DeviceVector_array1d;
	typedef cusp::array1d<float, cusp::host_memory>   HostVector_array1d;

	typedef cusp::csr_matrix<IndexType, ValueType, cusp::host_memory>   HostMatrix_csr;
	typedef cusp::csr_matrix<IndexType, ValueType, cusp::device_memory>   DeviceMatrix_csr;

	typedef lambda::composite_matrix<IndexType, ValueType, cusp::host_memory, HostMatrix_csr>   HostMatrix_comp;
	typedef lambda::composite_matrix<IndexType, ValueType, cusp::device_memory,  DeviceMatrix_csr> DeviceMatrix_comp;


private:
	DeviceMatrix_comp dev_mat;
	HostMatrix_comp host_mat;


public:

	void setUp()
	{

		culaStatus status;
		status = culaInitialize();
		checkStatus(status);

		std::string path = "data/positive-definite/lehmer20.mtx";


		HostMatrix_csr M11, M12, L11, L21, L22;
		cusp::io::read_matrix_market_file(M11, path);
		cusp::io::read_matrix_market_file(M12, path);
		cusp::io::read_matrix_market_file(L11, path);
		cusp::io::read_matrix_market_file(L21, path);
		cusp::io::read_matrix_market_file(L22, path);

		host_mat = HostMatrix_comp(M11,M12,L11,L21,L22);

		dev_mat = DeviceMatrix_comp(host_mat);

	}

	void tearDown()
	{
		culaShutdown();
	}

	void test_host_multiply(){

		HostVector_array1d y1(host_mat.num_rows);
		HostVector_array1d x(host_mat.num_cols, ValueType(1));
		cusp::multiply(host_mat,x,y1);


		// ******** TESTING *************

		// A = L11^{-1}*(M_{11}+M_{12}*L_{22}^{-1}*L_{21})
		HostMatrix_array2d L11_inv, L22_inv, L21, M11, M12;

		cusp::convert(host_mat.L11, L11_inv);
		cuspla::getri(L11_inv);
		cusp::convert(host_mat.L22, L22_inv);
		cuspla::getri(L22_inv);
		cusp::convert(host_mat.L21, L21);
		cusp::convert(host_mat.M11, M11);
		cusp::convert(host_mat.M12, M12);

		HostMatrix_array2d tmp1, tmp2;
		cuspla::gemm(M12, L22_inv, tmp1, ValueType(1));
		cuspla::gemm(tmp1, L21, tmp2, ValueType(1));

		cusp::blas::axpy(M11.values.begin() , M11.values.end(), tmp2.values.begin(), ValueType(1));
		HostMatrix_array2d A;
		cuspla::gemm(L11_inv, tmp2, A, ValueType(1));

		HostVector_array1d y2(host_mat.num_rows);
		cuspla::gemv(A,x,y2);

		ValueType errRel = nrmVector("host_multiply: ", y1, y2);
		CPPUNIT_ASSERT( errRel < 1.0e-3 );
	}

	void test_device_multiply(){

		DeviceVector_array1d y1(dev_mat.num_rows);
		DeviceVector_array1d x(dev_mat.num_cols, ValueType(1));
		cusp::multiply(dev_mat,x,y1);
		HostVector_array1d y1_host(host_mat.num_rows);
		cusp::copy(y1, y1_host);
		HostVector_array1d x_host(host_mat.num_rows);
		cusp::copy(x, x_host);



		// ******** TESTING *************

		// A = L11^{-1}*(M_{11}+M_{12}*L_{22}^{-1}*L_{21})
		HostMatrix_array2d L11_inv, L22_inv, L21, M11, M12;

		cusp::convert(host_mat.L11, L11_inv);
		cuspla::getri(L11_inv);
		cusp::convert(host_mat.L22, L22_inv);
		cuspla::getri(L22_inv);
		cusp::convert(host_mat.L21, L21);
		cusp::convert(host_mat.M11, M11);
		cusp::convert(host_mat.M12, M12);

		HostMatrix_array2d tmp1, tmp2;
		cuspla::gemm(M12, L22_inv, tmp1, ValueType(1));
		cuspla::gemm(tmp1, L21, tmp2, ValueType(1));

		cusp::blas::axpy(M11.values.begin() , M11.values.end(), tmp2.values.begin(), ValueType(1));
		HostMatrix_array2d A;
		cuspla::gemm(L11_inv, tmp2, A, ValueType(1));

		HostVector_array1d y2(host_mat.num_rows);
		cuspla::gemv(A,x_host,y2);


		ValueType errRel = nrmVector("device_multiply: ", y1_host, y2);
		CPPUNIT_ASSERT( errRel < 1.0e-3 );
	}


	void test_host_arnoldi()
	{

		size_t m = 10;
		HostMatrix_array2d H(m, m);
		HostMatrix_array2d V(host_mat.num_rows, m);
		HostVector_array1d f(host_mat.num_rows, ValueType(0));

		cusp::krylov::arnoldi(host_mat, H, V, f, 0, 3);
		cusp::krylov::arnoldi(host_mat, H, V, f, 2, 5);
		cusp::krylov::arnoldi(host_mat, H, V, f, 4, m);



		// ******* TESTING ***********

		HostMatrix_array2d A2d;
		HostMatrix_array2d V2;
		HostMatrix_array2d H2;

		HostMatrix_array2d C;
		HostMatrix_array2d C2;


		size_t N = host_mat.num_rows;

		// A = L11^{-1}*(M_{11}+M_{12}*L_{22}^{-1}*L_{21})
		HostMatrix_array2d L11_inv, L22_inv, L21, M11, M12;

		cusp::convert(host_mat.L11, L11_inv);
		cuspla::getri(L11_inv);
		cusp::convert(host_mat.L22, L22_inv);
		cuspla::getri(L22_inv);
		cusp::convert(host_mat.L21, L21);
		cusp::convert(host_mat.M11, M11);
		cusp::convert(host_mat.M12, M12);

		HostMatrix_array2d tmp1, tmp2;
		cuspla::gemm(M12, L22_inv, tmp1, ValueType(1));
		cuspla::gemm(tmp1, L21, tmp2, ValueType(1));

		cusp::blas::axpy(M11.values.begin() , M11.values.end(), tmp2.values.begin(), ValueType(1));
		cuspla::gemm(L11_inv, tmp2, A2d, ValueType(1));


		// create submatrix V2
		cusp::copy(V, V2);
		V2.resize(N,m);

		// create submatrix H2
		H2.resize(m,m);
		size_t l = H.num_rows;
		for(size_t j=0; j<m; j++)
			thrust::copy(H.values.begin()+ l*j, H.values.begin()+ l*j +m, H2.values.begin()+ m*j);

		cusp::multiply(A2d, V2, C);

		cusp::multiply(V2, H2, C2);

		cusp::blas::axpy(f.begin() , f.end(), C2.values.begin()+(m-1)*N, ValueType(1));


		ValueType errRel = nrmVector("host_arnoldi: ", C.values, C2.values);
		CPPUNIT_ASSERT( errRel < 1.0e-3 );

	}

	void test_device_arnoldi()
	{

		size_t m = 10;
		DeviceMatrix_array2d H(m, m);
		DeviceMatrix_array2d V(dev_mat.num_rows, m);
		DeviceVector_array1d f(dev_mat.num_rows, ValueType(0));

		//		  DeviceMatrix_csr dev_mat;
		//		  cusp::convert(dev_mat_def_pos[i], dev_mat);
		cusp::krylov::arnoldi(dev_mat, H, V, f, 0, 3);
		cusp::krylov::arnoldi(dev_mat, H, V, f, 2, 5);
		cusp::krylov::arnoldi(dev_mat, H, V, f, 4, m);


		// ******* TESTING ***********

		HostMatrix_array2d A2d;
		HostMatrix_array2d V2;
		HostMatrix_array2d H2;

		HostMatrix_array2d C;
		HostMatrix_array2d C2;
		HostVector_array1d f_host;
		cusp::convert(f,f_host);

		size_t N = host_mat.num_rows;

		// A = L11^{-1}*(M_{11}+M_{12}*L_{22}^{-1}*L_{21})
		HostMatrix_array2d L11_inv, L22_inv, L21, M11, M12;

		cusp::convert(host_mat.L11, L11_inv);
		cuspla::getri(L11_inv);
		cusp::convert(host_mat.L22, L22_inv);
		cuspla::getri(L22_inv);
		cusp::convert(host_mat.L21, L21);
		cusp::convert(host_mat.M11, M11);
		cusp::convert(host_mat.M12, M12);

		HostMatrix_array2d tmp1, tmp2;
		cuspla::gemm(M12, L22_inv, tmp1, ValueType(1));
		cuspla::gemm(tmp1, L21, tmp2, ValueType(1));

		cusp::blas::axpy(M11.values.begin() , M11.values.end(), tmp2.values.begin(), ValueType(1));
		cuspla::gemm(L11_inv, tmp2, A2d, ValueType(1));



		// create submatrix V2
		cusp::copy(V, V2);
		V2.resize(N,m);

		// create submatrix H2
		H2.resize(m,m);
		size_t l = H.num_rows;
		for(size_t j=0; j<m; j++)
			thrust::copy(H.values.begin()+ l*j, H.values.begin()+ l*j +m, H2.values.begin()+ m*j);

		cusp::multiply(A2d, V2, C);

		cusp::multiply(V2, H2, C2);

		cusp::blas::axpy(f_host.begin() , f_host.end(), C2.values.begin()+(m-1)*N, float(1));



		ValueType errRel = nrmVector("device_arnoldi: ", C.values, C2.values);
		CPPUNIT_ASSERT( errRel < 1.0e-3 );

	}


	void test_host_iram(){ //  TODO test iram with composite matrix
		size_t k = 4;

		size_t n = host_mat.num_rows;
		size_t m = host_mat.num_cols;
		HostMatrix_array2d eigvects;
		HostMatrix_array2d A2d;
		HostVector_array1d eigvals;
		HostVector_array1d y1, eigvec(m);

		cusp::krylov::implicitly_restarted_arnoldi(host_mat,\
				eigvals, eigvects, k, 0);

		// A = L11^{-1}*(M_{11}+M_{12}*L_{22}^{-1}*L_{21})
		HostMatrix_array2d L11_inv, L22_inv, L21, M11, M12;

		cusp::convert(host_mat.L11, L11_inv);
		cuspla::getri(L11_inv);
		cusp::convert(host_mat.L22, L22_inv);
		cuspla::getri(L22_inv);
		cusp::convert(host_mat.L21, L21);
		cusp::convert(host_mat.M11, M11);
		cusp::convert(host_mat.M12, M12);

		HostMatrix_array2d tmp1, tmp2;
		cuspla::gemm(M12, L22_inv, tmp1, ValueType(1));
		cuspla::gemm(tmp1, L21, tmp2, ValueType(1));

		cusp::blas::axpy(M11.values.begin() , M11.values.end(), tmp2.values.begin(), ValueType(1));
		cuspla::gemm(L11_inv, tmp2, A2d, ValueType(1));


		for(size_t j=0; j<eigvals.size(); j++){
			thrust::copy(eigvects.values.begin()+ j*n, eigvects.values.begin()+ (j+1)*n,eigvec.begin());
			cuspla::gemv(A2d, eigvec, y1, false);
			cusp::blas::scal(eigvec, (ValueType)eigvals[j]);

			std::stringstream j_str, eigval_str;
			j_str << j;
			eigval_str << eigvals[j];

			ValueType errRel = nrmVector("host_iram eigval["+j_str.str()+"]:"+eigval_str.str(), y1, eigvec);
			CPPUNIT_ASSERT( errRel < 1.0e-2 );
		}
	}


//	test_device_iram(){
//		size_t k = 4;
//
//		size_t n = host_mat.num_rows;
//		size_t m = host_mat.num_cols;
//		DeviceMatrix_array2d eigvects;
//		HostMatrix_array2d A2d;
//		DeviceVector_array1d eigvals;
//		HostVector_array1d y1, eigvec(m);
//
//		cusp::krylov::implicitly_restarted_arnoldi(dev_mat,\
//				eigvals, eigvects, k, 0);
//
//
//		cusp::convert(dev_mat, A2d);
//
//		for(size_t j=0; j<eigvals.size(); j++){
//			thrust::copy(eigvects.values.begin()+ j*n, eigvects.values.begin()+ (j+1)*n,eigvec.begin());
//			cuspla::gemv(A2d, eigvec, y1, false);
//			cusp::blas::scal(eigvec, (ValueType)eigvals[j]);
//
//			std::stringstream j_str, eigval_str;
//			j_str << j;
//			eigval_str << eigvals[j];
//
//			ValueType errRel = nrmVector("host_iram eigval["+j_str.str()+"]:"+eigval_str.str(), y1, eigvec);
//			CPPUNIT_ASSERT( errRel < 1.0e-2 );
//
//		}
//	}



	template <typename Array1d>
	ValueType nrmVector(std::string title, Array1d& A, Array1d& A2){
		ValueType nrmA = cusp::blas::nrm2(A);
		ValueType nrmA2 = cusp::blas::nrm2(A2);
		// Calculates the difference and overwrite the matrix C
		cusp::blas::axpy(A, A2, ValueType(-1));
		ValueType nrmDiff = cusp::blas::nrm2(A2);



		ValueType errRel = ValueType(0);
		if(nrmA==ValueType(0))
			errRel = ValueType(1.0e-30);
		else
			errRel = nrmDiff/nrmA;

#ifdef VERBOSE
#ifndef VVERBOSE
		if(errRel != errRel || errRel >= 1.0e-2){ // Checks if error is nan
#endif VVERBOSE

			std::cout << title << ": AbsoluteErr=" << nrmDiff <<\
					" RelativeErr=" << errRel << "\n" << std::endl;
#ifndef VVERBOSE
		}
#endif VVERBOSE
#endif


		return errRel;
	}



};





CPPUNIT_TEST_SUITE_REGISTRATION( LambdaTestCase );

int main(int argc, char** argv)
{

	CppUnit::TextUi::TestRunner runner;
	CppUnit::TestFactoryRegistry &registry = CppUnit::TestFactoryRegistry::getRegistry();
	runner.addTest( registry.makeTest() );
	runner.run();
	return 0;

}



